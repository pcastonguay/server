#include "hip/hip_runtime.h"
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "src/core/kernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define THREADBLOCK_SIZE 512
__launch_bounds__(THREADBLOCK_SIZE)
__global__ void tritonBytesizeGatherKernel(
    const int8_t ** __restrict input_ptr_buffer,
    const size_t * __restrict byte_size_buffer,
    const size_t * __restrict byte_size_offset_buffer,
    int8_t * __restrict output_buffer)
{
    int request_idx = blockIdx.x;
    int laneId = threadIdx.x;
    const int8_t * request_input_buffer = input_ptr_buffer[request_idx];
    int byte_size = byte_size_buffer[request_idx];
    int byte_size_offset = byte_size_offset_buffer[request_idx];

    int8_t * output_buffer_with_offset = output_buffer + byte_size_offset;
    for(int elemId = laneId; elemId < byte_size; elemId += THREADBLOCK_SIZE)
    {
        output_buffer_with_offset[elemId] = __ldg(request_input_buffer + elemId);
    }
}

void runGatherKernel(
    const int8_t ** input_ptr_buffer,
    const size_t * byte_size_buffer,
    const size_t * byte_size_offset_buffer,
    int8_t * output_buffer,
    size_t request_count,
    hipStream_t stream)
{
    tritonBytesizeGatherKernel<<<request_count,THREADBLOCK_SIZE,0,stream>>>(
        input_ptr_buffer,
        byte_size_buffer,
        byte_size_offset_buffer,
        output_buffer);
}